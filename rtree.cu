#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include "rtree.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>    
#include <assert.h>
#include <math.h>
#include <linux/hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>

using namespace hipcub; // debug

/// \todo fix to not be global
CachingDeviceAllocator g_allocator(true); // CUB caching allocator for device memory

inline void update_boundary(struct rtree_rect* boundary, struct rtree_point* p) {
  /// \todo replace these with CUDA min/max which won't use conditionals
  boundary->top = fmin(p->y, boundary->top);
  boundary->bottom = fmax(p->y, boundary->bottom);
  boundary->left = fmin(p->x, boundary->left);
  boundary->right = fmax(p->x, boundary->right);
}

inline void update_boundary(struct rtree_rect* boundary, struct rtree_rect* node) {
  /// \todo replace these with CUDA min/max which won't use conditionals
  boundary->top = fmin(node->top, boundary->top);
  boundary->bottom = fmax(node->bottom, boundary->bottom);
  boundary->left = fmin(node->left, boundary->left);
  boundary->right = fmax(node->right, boundary->right);
}

/// initialize boundary so the first udpate overrides it.
inline void init_boundary(struct rtree_rect* boundary) {
  boundary->top = ord_t_max;
  boundary->bottom = ord_t_lowest;
  boundary->left = ord_t_max;
  boundary->right = ord_t_lowest;
}

/// used to calculate tree height
/// \todo use CUDA maths
inline size_t log_base_ceil(const size_t x, const size_t base) {
  return (size_t)ceil(log((double)x) / log((double)base));
}


inline size_t get_node_length(const size_t i, const size_t level_len, const size_t previous_level_len, const size_t node_size) {
  // let would be nice.
  const size_t n = node_size;
  const size_t len = previous_level_len;
  const size_t final_i = level_len - 1; // this better be optimised out
  // this nasty bit sets lnum to len % n if it's the last iteration and there's a remainder, else n
  // which avoids a GPU-breaking conditional
  return ((i != final_i || len % n == 0) * n) + ((i == final_i && len % n != 0) * (len % n));
}

struct rtree cuda_create_rtree(struct rtree_points points) {
//  struct rtree tree;
//  tree.depth = log_base_ceil(points.length, RTREE_NODE_SIZE);
//  tree.levels = (rtree_node**) malloc(sizeof(struct rtree_node) * tree.depth);

  struct rtree_leaf* leaves = cuda_create_leaves(cuda_sort(points));
  const size_t leaves_len = DIV_CEIL(points.length, RTREE_NODE_SIZE);

  // points can be deleted now;

/*  
  tree.levels[0] = (rtree_node*) leaves;
  for(size_t i = 1, end = tree.depth, previous_len = leaves_len; i != end; ++i, previous_len = DIV_CEIL(previous_len, RTREE_NODE_SIZE)) {
    tree.levels[i] = cuda_create_level(tree.levels[i - 1], previous_len);
  }
*/

  rtree_node* previous_level = (rtree_node*) leaves;
  size_t      previous_len = leaves_len;
  size_t      depth = 1; // leaf level is 0
  while(previous_len > RTREE_NODE_SIZE) {
    previous_level = cuda_create_level(previous_level, previous_len);
    previous_len = DIV_CEIL(previous_len, RTREE_NODE_SIZE);
    ++depth;
  }

  rtree_node* root = (rtree_node*) malloc(sizeof(rtree_node));
  init_boundary(&root->bounding_box);
  root->num = previous_len;
  root->children = previous_level;
  for(size_t i = 0, end = previous_len; i != end; ++i)
    update_boundary(&root->bounding_box, &root->children[i].bounding_box);
  ++depth;

  struct rtree tree = {depth, root};
  return tree;
}

/// \param nodes Can really be either a rtree_node or rtree_leaf; doesn't matter to us, we won't dereference .children
/// \return next level up. Length is ceil(nodes_len / RTREE_NODE_SIZE)
struct rtree_node* cuda_create_level(struct rtree_node* nodes, const size_t nodes_len) {
  const size_t next_level_len = DIV_CEIL(nodes_len, RTREE_NODE_SIZE);
  rtree_node* next_level = (rtree_node*) malloc(sizeof(rtree_node) * next_level_len);
  init_boundary(&next_level->bounding_box);

  for(size_t i = 0, end = next_level_len; i != end; ++i) {
    rtree_node* n = &next_level[i];
    init_boundary(&n->bounding_box);
    n->num = get_node_length(i, end, next_level_len, RTREE_NODE_SIZE);
    // n->nodes doesn't need malloced - it will point to the node in nodes
    n->children = (rtree_node*)&nodes[i * RTREE_NODE_SIZE];

#   pragma unroll
    for(size_t j = 0, jend = n->num; j != jend; ++j)
      update_boundary(&n->bounding_box, &n->children[j].bounding_box);
  }
  return next_level;
}

/// \todo make CUDA
struct rtree_leaf* cuda_create_leaves(struct rtree_points sorted) {
  static_assert(sizeof(rtree_node) == sizeof(rtree_leaf), "rtree node, leaf sizes must be equal, since leaves are passed to create_level");

  struct rtree_leaf* leaves = (rtree_leaf*) malloc(sizeof(rtree_leaf) * sorted.length);
  for(size_t i = 0, end = DIV_CEIL(sorted.length, RTREE_NODE_SIZE); i != end; ++i) {
    rtree_leaf* l = &leaves[i];
    init_boundary(&l->bounding_box);
    l->num = get_node_length(i, end, sorted.length, RTREE_NODE_SIZE);
    l->points = (rtree_point*) malloc(sizeof(rtree_point) * l->num);
    
#   pragma unroll
    for(size_t j = 0, jend = l->num; j != jend; ++j) {
      rtree_point* p = &l->points[j];
      p->x   = sorted.x[i * RTREE_NODE_SIZE + j];
      p->y   = sorted.ykey[i * RTREE_NODE_SIZE + j].y;
      p->key = sorted.ykey[i * RTREE_NODE_SIZE + j].key;
      update_boundary(&l->bounding_box, p);
    }
  }
  return leaves;
}

struct rtree_points cuda_sort(struct rtree_points points) {
  typedef ord_t key_t;
  typedef struct rtree_y_key value_t;
  DoubleBuffer<key_t> d_keys;
  DoubleBuffer<value_t> d_values;
  CubDebugExit( g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(key_t) * points.length));
  CubDebugExit( g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(key_t) * points.length));
  CubDebugExit( g_allocator.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(value_t) * points.length));
  CubDebugExit( g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(value_t) * points.length));

  CubDebugExit( hipMemcpy(d_keys.d_buffers[0], points.x, sizeof(key_t) * points.length, hipMemcpyHostToDevice));
  CubDebugExit( hipMemcpy(d_values.d_buffers[0], points.ykey, sizeof(value_t) * points.length, hipMemcpyHostToDevice));

  size_t temp_storage_bytes = 0;
  void* d_temp_storage = NULL;
  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, points.length));
  CubDebugExit( g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, points.length));
  
  CubDebugExit( hipMemcpy(points.x, d_keys.Current(), points.length * sizeof(key_t), hipMemcpyDeviceToHost));
  CubDebugExit( hipMemcpy(points.ykey, d_values.Current(), points.length * sizeof(value_t), hipMemcpyDeviceToHost));

  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_temp_storage));

  return points;
}

/// \returns the device totalGlobalMem
inline size_t GetDeviceMemory() {
  hipDeviceProp_t properties;
  int deviceNum;
  CubDebugExit(hipGetDevice(&deviceNum));
  CubDebugExit(hipGetDeviceProperties(&properties, deviceNum));
  return properties.totalGlobalMem;
}

#ifndef MIN
#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#endif

inline size_t find_min(location_t* keys, const size_t keys_len) {
  if(keys_len == 0)
    return 0;
  location_t min = keys[0];
  size_t min_key = 0;
  for(size_t i = 0, end = keys_len; i != end; ++i) {
    if(keys[i] < min) {
      min_key = i;
      min = keys[i];
    }
  }
  return min_key;
}

/// \param[out] keys must be at least block_len large
/// \return whether all iterators are past their length. That is, when this is false, we can stop merging.
inline bool get_keys(location_t* keys, const struct linear_quadtree* array_blocks, const size_t block_len, const size_t* iterators) {
  bool got_key = false;
  for(int i = 0, end = block_len; i != end; ++i) {
    if(iterators[i] >= array_blocks[i].length) {
      keys[i] = location_t_max; // we've iterated past this block's len; make sure this key is never the min.
      continue;
    }
    got_key = true;
    keys[i] = array_blocks[i].locations[iterators[i]];
  }
  return got_key;
}

struct linear_quadtree lqt_merge(struct linear_quadtree* array_blocks, const size_t block_len, struct rtree_point* points, const size_t len) {
  struct linear_quadtree lqt;
  lqt.points    = points;
  lqt.locations = (location_t*) malloc(sizeof(location_t) * len);
  lqt.length    = len;
  if(len == 0)
    return lqt;

  size_t lqt_iterator = 0;
  size_t* iterators = (size_t*) malloc(sizeof(size_t) * block_len);
  for(size_t i = 0, end = block_len; i != end; ++i)
    iterators[i] = 0;

  {
    location_t keys[block_len];  
    for(size_t i = 0; get_keys(keys, array_blocks, block_len, iterators); ++i) {
      const size_t min_block = find_min(keys, block_len);
      lqt.locations[lqt_iterator] = array_blocks[min_block].locations[iterators[min_block]];
      lqt.points[lqt_iterator]    = array_blocks[min_block].points[iterators[min_block]];
      ++iterators[min_block];
      ++lqt_iterator;
    }
  }
  
  free(iterators);
  return lqt;
}

__global__ void nodify_kernel(struct rtree_point* points, location_t* locations,
                                 const size_t depth, ord_t xstart, ord_t xend, 
                                 ord_t ystart, ord_t yend, size_t len) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i >= len)
    return; // skip the final block remainder

  struct rtree_point* thisPoint = &points[i];

  ord_t currentXStart = xstart;
  ord_t currentXEnd = xend;
  ord_t currentYStart = ystart;
  ord_t currentYEnd = yend;
  for(size_t j = 0, jend = depth; j != jend; ++j) {
    const location_t bit1 = thisPoint->y > (currentYStart + (currentYEnd - currentYStart) / 2);
    const location_t bit2 = thisPoint->x > (currentXStart + (currentXEnd - currentXStart) / 2);
    const location_t currentPosBits = (bit1 << 1) | bit2;
    locations[i] = (locations[i] << 2) | currentPosBits;

    const ord_t newWidth = (currentXEnd - currentXStart) / 2;
    currentXStart = floor((thisPoint->x - currentXStart) / newWidth) * newWidth + currentXStart;
    currentXEnd = currentXStart + newWidth;
    const ord_t newHeight = (currentYEnd - currentYStart) / 2;
    currentYStart = floor((thisPoint->y - currentYStart) / newHeight) * newHeight + currentYStart;
    currentYEnd = currentYStart + newHeight;
  }
}

struct linear_quadtree lqt_create_cuda(struct rtree_point* points, size_t len, 
                                       ord_t xstart, ord_t xend, 
                                       ord_t ystart, ord_t yend,
                                       size_t* depth) {
  // debug
  size_t cuda_mem_free = 0;
  size_t cuda_mem_total = 0;
  CubDebugExit(hipMemGetInfo(&cuda_mem_free, &cuda_mem_total));
  cuda_mem_free = cuda_mem_free / 5 * 4; // wiggle room <('.'<) <('.' )> (>'.')>

  const size_t array_size = (sizeof(struct rtree_point) + sizeof(location_t)) * len * 2; // *2 for double-buffers
  const size_t num_blocks = array_size / cuda_mem_free + 1;
  printf("num blocks: %lu\n", num_blocks); // debug
  const size_t array_block_size = array_size / num_blocks;
  printf("free: %lu\tarray: %lu\tblocks: %lu\tblock size: %lu\n", cuda_mem_free, array_size, num_blocks, array_block_size); // debug
  
  const size_t block_len = len / num_blocks + (len % num_blocks != 0 ? 1 : 0);
  struct linear_quadtree* array_blocks = (struct linear_quadtree*) malloc(num_blocks * sizeof(linear_quadtree));

  for(size_t i = 0, end = num_blocks; i != end; ++i) {
    array_blocks[i].length = block_len;
    if(block_len * i + block_len  > len)
      array_blocks[i].length -= block_len * num_blocks - len; // fix the last block overlap
    array_blocks[i].points = (struct rtree_point*) malloc(sizeof(struct rtree_point) * array_blocks[i].length);
    memcpy(array_blocks[i].points, points + block_len * i, array_blocks[i].length * sizeof(struct rtree_point));
    array_blocks[i] = lqt_sortify_cuda_mem(lqt_nodify_cuda_mem(array_blocks[i].points, array_blocks[i].length, xstart, xend, ystart, yend, depth));
  }
  
  struct linear_quadtree lqt = lqt_merge(array_blocks, num_blocks, points, len);
  for(size_t i = 0, end = num_blocks; i != end; ++i)
    lqt_delete(array_blocks[i]);
  free(array_blocks);
  return lqt;
}

/// unnecessarily allocates and frees CUDA memory twice
struct linear_quadtree lqt_create_cuda_slow(struct rtree_point* points, size_t len, 
                                       ord_t xstart, ord_t xend, 
                                       ord_t ystart, ord_t yend,
                                       size_t* depth) {
  return lqt_sortify_cuda(lqt_nodify_cuda(points, len, xstart, xend, ystart, yend, depth));
}


struct linear_quadtree lqt_nodify_cuda(struct rtree_point* points, size_t len, 
                                       ord_t xstart, ord_t xend, 
                                       ord_t ystart, ord_t yend,
                                       size_t* depth) {
  *depth = LINEAR_QUADTREE_DEPTH;

  const size_t THREADS_PER_BLOCK = 512;

  location_t*       cuda_locations;
  struct rtree_point* cuda_points;

  hipMalloc((void**)&cuda_locations, len * sizeof(location_t));
  hipMalloc((void**)&cuda_points, len * sizeof(struct rtree_point));
  hipMemcpy(cuda_points, points, len * sizeof(struct rtree_point), hipMemcpyHostToDevice);
  hipMemset(cuda_locations, 0, len * sizeof(location_t)); // debug
  nodify_kernel<<<(len + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(cuda_points, cuda_locations, *depth, xstart, xend, ystart, yend, len);
  location_t* locations = (location_t*) malloc(len * sizeof(location_t));
  hipMemcpy(locations, cuda_locations, len * sizeof(location_t), hipMemcpyDeviceToHost);
  hipFree(cuda_locations);
  hipFree(cuda_points);

  struct linear_quadtree lqt;
  lqt.points    = points;
  lqt.locations = locations;
  lqt.length    = len;
  return lqt;
}

struct linear_quadtree lqt_sortify_cuda(struct linear_quadtree lqt) {
  DoubleBuffer<location_t> d_keys;
  DoubleBuffer<rtree_point> d_values;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(location_t) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(location_t) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(rtree_point) * lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(rtree_point) * lqt.length));


  CubDebugExit( hipMemcpy(d_keys.d_buffers[0], lqt.locations, sizeof(location_t) * lqt.length, hipMemcpyHostToDevice));
  CubDebugExit( hipMemcpy(d_values.d_buffers[0], lqt.points, sizeof(rtree_point) * lqt.length, hipMemcpyHostToDevice));

  size_t temp_storage_bytes = 0;
  void* d_temp_storage = NULL;
  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, lqt.length));
  CubDebugExit( g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, lqt.length));
  
  CubDebugExit( hipMemcpy(lqt.locations, d_keys.Current(), lqt.length * sizeof(location_t), hipMemcpyDeviceToHost));
  CubDebugExit( hipMemcpy(lqt.points, d_values.Current(), lqt.length * sizeof(rtree_point), hipMemcpyDeviceToHost));

  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_temp_storage));
  return lqt;
}

void print_array_uint(unsigned int* array, const size_t len) {
  if(len == 0)
    return;
  printf("[%u", array[0]);
  for(size_t i = 1, end = len; i != end; ++i)
    printf(" %u", array[i]);
  printf("]");
}
void print_array_int(int* array, const size_t len) {
  if(len == 0)
    return;
  printf("[%d", array[0]);
  for(size_t i = 1, end = len; i != end; ++i)
    printf(" %d", array[i]);
  printf("]");
}

template <typename T> struct fmt_traits;
template <>
struct fmt_traits<int> {
  static const char* str() {return "%d";}
};
template <>
struct fmt_traits<unsigned int> {
  static const char* str() {return "%u";}
};
template <>
struct fmt_traits<location_t> {
  static const char* str() {return "%lu";}
};

template <typename T>
void print_array(T* array, const size_t len) {
  if(len == 0)
    return;
  printf("[");
  printf(fmt_traits<T>::str(), array[0]);
  for(size_t i = 1, end = len; i != end; ++i) {
    printf(" ");
    printf(fmt_traits<T>::str(), array[i]);
  }
  printf("]");
}

// @return CUDA-allocated points and locations, along with existing host-allocated points
struct linear_quadtree_cuda lqt_nodify_cuda_mem(struct rtree_point* points, size_t len, 
                                                ord_t xstart, ord_t xend, 
                                                ord_t ystart, ord_t yend,
                                                size_t* depth) {
  const size_t THREADS_PER_BLOCK = 512;
  *depth = LINEAR_QUADTREE_DEPTH;
  location_t*       cuda_locations;
  struct rtree_point* cuda_points;

  CubDebugExit(g_allocator.DeviceAllocate((void**)&cuda_locations, sizeof(location_t) * len));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&cuda_points, sizeof(rtree_point) * len));
//  hipMalloc((void**)&cuda_locations, len * sizeof(location_t));
//  hipMalloc((void**)&cuda_points, len * sizeof(struct rtree_point));
  hipMemcpy(cuda_points, points, len * sizeof(struct rtree_point), hipMemcpyHostToDevice);
  hipMemset(cuda_locations, 0, len * sizeof(location_t)); // debug
  nodify_kernel<<<(len + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(cuda_points, cuda_locations, *depth, xstart, xend, ystart, yend, len);

  struct linear_quadtree_cuda lqt;
  lqt.points         = points;
  lqt.cuda_locations = cuda_locations;
  lqt.cuda_points    = cuda_points;
  lqt.length         = len;
  return lqt;
}

struct linear_quadtree lqt_sortify_cuda_mem(struct linear_quadtree_cuda cuda_lqt) {
  //  printf("DEBUG lqt_sortify_cuda_mem\n"); // debug

  DoubleBuffer<location_t> d_keys;
  DoubleBuffer<rtree_point> d_values;
  d_keys.d_buffers[0]   = cuda_lqt.cuda_locations; // reuse the nodify CUDA memory for the cub buffers
  d_values.d_buffers[0] = cuda_lqt.cuda_points;
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(location_t) * cuda_lqt.length));
  CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(rtree_point) * cuda_lqt.length));

  size_t temp_storage_bytes = 0;
  void* d_temp_storage = NULL;
  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, cuda_lqt.length));
  //  printf("temp storage: %lu\n", temp_storage_bytes);  // debug
  CubDebugExit( g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

  CubDebugExit( DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, cuda_lqt.length));

  struct linear_quadtree lqt;
  lqt.length = cuda_lqt.length;
  lqt.locations = (location_t*) malloc(lqt.length * sizeof(location_t));
  CubDebugExit( hipMemcpy(lqt.locations, d_keys.Current(), lqt.length * sizeof(location_t), hipMemcpyDeviceToHost));
  lqt.points = cuda_lqt.points;
  CubDebugExit( hipMemcpy(lqt.points, d_values.Current(), lqt.length * sizeof(rtree_point), hipMemcpyDeviceToHost));

  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_keys.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[0]));
  CubDebugExit( g_allocator.DeviceFree(d_values.d_buffers[1]));
  CubDebugExit( g_allocator.DeviceFree(d_temp_storage));
  return lqt;
}

